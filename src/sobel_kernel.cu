#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <iostream>

__global__ void sobel_kernel(
    const unsigned char* input,
    unsigned char* output,
    int width,
    int height,
    int input_step)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Skip border pixels
    if (x <= 0 || y <= 0 || x >= width - 1 || y >= height - 1)
        return;

    int offset = y * input_step + x;

    // Sobel X
    int Gx =
        -1 * input[(y - 1) * input_step + (x - 1)] +
         0 * input[(y - 1) * input_step + (x    )] +
         1 * input[(y - 1) * input_step + (x + 1)] +
        -2 * input[(y    ) * input_step + (x - 1)] +
         0 * input[(y    ) * input_step + (x    )] +
         2 * input[(y    ) * input_step + (x + 1)] +
        -1 * input[(y + 1) * input_step + (x - 1)] +
         0 * input[(y + 1) * input_step + (x    )] +
         1 * input[(y + 1) * input_step + (x + 1)];

    // Sobel Y
    int Gy =
        -1 * input[(y - 1) * input_step + (x - 1)] +
        -2 * input[(y - 1) * input_step + (x    )] +
        -1 * input[(y - 1) * input_step + (x + 1)] +
         0 * input[(y    ) * input_step + (x - 1)] +
         0 * input[(y    ) * input_step + (x    )] +
         0 * input[(y    ) * input_step + (x + 1)] +
         1 * input[(y + 1) * input_step + (x - 1)] +
         2 * input[(y + 1) * input_step + (x    )] +
         1 * input[(y + 1) * input_step + (x + 1)];

    int magnitude = abs(Gx) + abs(Gy);  // approximation

    // Clamp to [0, 255]
    magnitude = min(255, magnitude);

    output[offset] = static_cast<unsigned char>(magnitude);
}


cv::Mat apply_gpu_sobel(const cv::Mat& input_gray) 
{
    CV_Assert(input_gray.type() == CV_8UC1);

    int width = input_gray.cols;
    int height = input_gray.rows;
    size_t input_bytes = input_gray.step * height;
    size_t output_bytes = width * height;

    cv::Mat output(height, width, CV_8UC1);

    unsigned char *d_input = nullptr, *d_output = nullptr;
    hipError_t err;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate device memory
    err = hipMalloc(&d_input, input_bytes);
    if (err != hipSuccess) 
    {
        std::cerr << "hipMalloc d_input failed: " << hipGetErrorString(err) << "\n";
        std::exit(EXIT_FAILURE);
    }

    err = hipMalloc(&d_output, output_bytes);
    if (err != hipSuccess) 
    {
        std::cerr << "hipMalloc d_output failed: " << hipGetErrorString(err) << "\n";
        std::exit(EXIT_FAILURE);
    }

    // Copy input to device
    err = hipMemcpy(d_input, input_gray.ptr(), input_bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) 
    {
        std::cerr << "hipMemcpy H2D failed: " << hipGetErrorString(err) << "\n";
        std::exit(EXIT_FAILURE);
    }

    // Launch kernel
    dim3 blockDim(16, 16);
    dim3 gridDim((width + 15) / 16, (height + 15) / 16);
    hipEventRecord(start);
    sobel_kernel<<<gridDim, blockDim>>>(d_input, d_output, width, height, input_gray.step);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Check for launch errors
    err = hipGetLastError();
    if (err != hipSuccess) 
    {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << "\n";
        std::exit(EXIT_FAILURE);
    }

    // Wait for kernel to finish
    err = hipDeviceSynchronize();
    if (err != hipSuccess) 
    {
        std::cerr << "hipDeviceSynchronize failed: " << hipGetErrorString(err) << "\n";
        std::exit(EXIT_FAILURE);
    }
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "GPU kernel time: " << milliseconds << " ms" << std::endl;


    // Copy output back
    err = hipMemcpy(output.ptr(), d_output, output_bytes, hipMemcpyDeviceToHost);
    if (err != hipSuccess) 
    {
        std::cerr << "hipMemcpy D2H failed: " << hipGetErrorString(err) << "\n";
        std::exit(EXIT_FAILURE);
    }

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);

    return output;
}
