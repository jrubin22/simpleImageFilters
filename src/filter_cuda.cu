#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <iostream>

__global__ void grayscale_kernel(unsigned char* input, unsigned char* output, int width, int height, int channels) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int idx = (y * width + x) * channels;

    if (x < width && y < height) 
    {
        unsigned char r = input[idx + 2];
        unsigned char g = input[idx + 1];
        unsigned char b = input[idx + 0];
        output[y * width + x] = static_cast<unsigned char>(0.21f * r + 0.72f * g + 0.07f * b);
    }
}

cv::Mat apply_gpu_grayscale(const cv::Mat& input) 
{
    hipError_t err;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    CV_Assert(input.channels() == 3);
    int width = input.cols;
    int height = input.rows;
    int channels = input.channels();
    size_t input_bytes = width * height * channels;
    size_t output_bytes = width * height;

    // Allocate host output
    cv::Mat output(height, width, CV_8UC1);

    // Allocate device memory
    unsigned char *d_input, *d_output;
    err = hipMalloc(&d_input, input_bytes);
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);

    }
    err = hipMalloc(&d_output, output_bytes);
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);

    }
    // Copy input to device
    err = hipMemcpy(d_input, input.ptr(), input_bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);

    }

    // Launch kernel
    dim3 blockDim(16, 16);
    dim3 gridDim((width + 15) / 16, (height + 15) / 16);
    hipEventRecord(start);
    grayscale_kernel<<<gridDim, blockDim>>>(d_input, d_output, width, height, channels);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    err = hipDeviceSynchronize();
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);

    }
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "GPU kernel time: " << milliseconds << " ms" << std::endl;


    // Copy output back to host
    err = hipMemcpy(output.ptr(), d_output, output_bytes, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);

    }
    // Free device memory
    err = hipFree(d_input);
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);

    }
    err = hipFree(d_output);
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);

    }
    hipEventDestroy(start);
    hipEventDestroy(stop);


    return output;
}
